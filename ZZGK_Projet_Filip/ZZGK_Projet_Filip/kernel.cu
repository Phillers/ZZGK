#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/functional.h>
#include<thrust/reduce.h>
#include <stdio.h>

const int size = 16;
const int bits = 4;
__global__ void joinGroups(int tab1[], int out[], int N) {
	extern __shared__ int tab[];
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < N) {
		tab[threadIdx.x] = tab1[idx];
	}

	_syncthreads();
	if (idx < N) {
		int a, b;
		if (threadIdx.x > 0) {
			a = tab[threadIdx.x - 1];
			b = tab[threadIdx.x];
		}
		else if (idx > 0) {
			a = tab1[idx - 1];
			b = tab[threadIdx.x];
		}
		int dif = a ^ b;
		if (__popc(dif) == 2&& dif > a&&dif > b) {
				out[idx] = 0;
		}
		else {
			out[idx] = 1;
		}
	}
}

struct isPositive {
	__host__ __device__ bool operator()(int x) {
		return x > 0;
	}
};

int main()
{
	thrust::host_vector<int> h_tab1;
	thrust::device_vector<int> d_tab1;
	thrust::device_vector<int> d_tab2;

	std::vector<int> sizes;
	int sum = 0;
	while(sum < size){
		int x = rand() % (size/2) + 1;
		if (sum + x > size)
			x = size - sum;
		sizes.push_back(x);
		sum += x;
	}
	int prefix = 0;
	int plength=0;
	for (int i = 1; i < bits;i++) {
		prefix <<= 1;
		prefix |= 1;
		plength += 1;
	}
	int dif = 1 << (plength-1);
	for (int x : sizes) {
		unsigned a = 1 << 31;
		for (int i = 0;i < x;i++) {
			h_tab1.push_back(a | prefix);
			a >>= 1;
		}
		prefix += dif;
		if (dif >> (plength - 1) == 1) {
			plength += 1;
		}
		dif >>= 1;
		if (dif == 0) {
			dif = 1 << (plength - 1);
		}
	}
	d_tab1 = h_tab1;
	d_tab2.resize(size);

	for (int i = 0;i < size;i++) {
		printf("%x\n", h_tab1[i]);
	}
		printf("\n\n");
	compare10 << <size / 64, 64, 64*sizeof(int) >> > (d_tab1.data().get(), d_tab2.data().get(), size);
	thrust::copy_if(thrust::c (0),
		thrust::make_counting_iterator(size),
		d_tab2.begin(),
		d_tab1.begin(),
		thrust::identity<int>());
	h_tab1 = d_tab1;
	for (int i = 0;i < size;i++) {
		printf("%d;", h_tab1[i]);
	}
	printf("\n\n");

	printf("%f", (h_tab1[size / 2] + h_tab1[size / 2 + 1]) / 2.0);
	*/
	return 0;
}

